
#include <hip/hip_runtime.h>
extern "C" __global__ void sum(const int N, const float *a, const float *b, float *c) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < N)
        c[i] = a[i] + b[i];
}
